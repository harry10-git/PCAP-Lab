#include<stdio.h>
#include "hip/hip_runtime.h"



__global__ void convolution(int *A, int *M, int *R, int N, int M_N)
{
    int tid = threadIdx.x;
    int start = tid -(M_N/2);
    int tot =0;

    for(int i=0; i<M_N; i++)
    {
        int pos = start + i;

        if(pos >= 0 && pos<N)
        {
            tot += A[pos]*M[i];
        }
    }
    printf("tot = %d\n", tot);

    R[tid] = tot;
}




int main(void)
{
    int A[7] = {1,2,3,4,5,6,7};
    int M[5] = {3,4,5,4,3};
    int R[7];

    int N= 7;
    int mask_N = 5;

    int *dA, *dM, *dR;

    hipMalloc((void**)&dA, N*sizeof(int));
    hipMalloc((void**)&dM, mask_N*sizeof(int));
    hipMalloc((void**)&dR, N*sizeof(int));

    hipMemcpy(dA, A, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dM, M, mask_N*sizeof(int), hipMemcpyHostToDevice);

    // kernel launch
    convolution<<<1,N>>>(dA,dM,dR,N,mask_N);

     hipMemcpy(R, dR, N*sizeof(int), hipMemcpyDeviceToHost);


    for(int i=0; i<N; i++)
    {
        printf("%d ", R[i]);
    }

    hipFree(dA);
    hipFree(dM);
    hipFree(dR);





}