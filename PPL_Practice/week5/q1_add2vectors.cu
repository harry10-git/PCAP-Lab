#include<stdio.h>
#include "hip/hip_runtime.h"



__global__ void vecAdd(int *A, int *B, int *C)
{
    int tid = threadIdx.x;
    C[tid] = A[tid]+ B[tid];
}

int main(void)
{
    int *A,*B,*C;
    int *dA,*dB, *dC;
    int n;
    printf("n: ");
    scanf("%d", &n);
    int size = n *sizeof(int);

    A = (int*)malloc(n*sizeof(int));
    B = (int*)malloc(n*sizeof(int));
    C = (int*)malloc(n*sizeof(int));


    printf("A: ");
    for(int i=0; i<n; i++)
    {
        scanf("%d", (A+i));
    }

    printf("B: ");
    for(int i=0; i<n; i++)
    {
        scanf("%d", (B+i));
    }

    hipMalloc((void**)&dA,size);
    hipMalloc((void**)&dB,size);
    hipMalloc((void**)&dC,size);

    hipMemcpy(dA,A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB,B, size, hipMemcpyHostToDevice);

    // launch kernel

    vecAdd<<<1,256>>>(dA,dB,dC);

    hipMemcpy(C,dC, size, hipMemcpyDeviceToHost);
    
    for(int i=0; i<n; i++)
    {
        printf("%d ",*(C+i));
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);


}

