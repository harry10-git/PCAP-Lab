#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void findA(char * str, int * d_count)
{
    int tid = threadIdx.x;
    
    if(str[tid] == 'a')
    {
        atomicAdd(d_count,1);
     
    }
}

int main(void)
{
    char str[25];
    char * d_str;

    int count=0 , *d_count;

    printf("string: ");
    scanf("%[^\n]s",str);

    hipMalloc((void**)&d_str, strlen(str)*sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_str, str, strlen(str)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    findA<<<1, strlen(str)>>>(d_str, d_count);

    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("num of A's = %d\n", count);

    hipFree(d_str);
    hipFree(d_count);

    return 0;
}