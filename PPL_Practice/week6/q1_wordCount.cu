#include "hip/hip_runtime.h"
// complete this

#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include<string.h>


__global__ void word_Counts(char * str, int * index, char * word, int * d_count)
{
    int tid = threadIdx.x;
    int word_len = strlen(word);

    int start = tid*2;
    int end = start +1;

    int len = start - end +1;

    if(len != word_len)
    {
        return;
    }

    bool flag = true;
    
    for(int i=0 ; i<word_len; i++)
    {
        if(str[start+i] != word[i])
        {
            flag = false;
            break;
        }
    }

    if(flag)
    {
        atomicAdd(d_count,1);
    }



}


int main(void)
{
    char str[100];

    scanf("%[^\n]s",str);


    int index[20];
    int ptr = 1;

    index[0] =0;

    for(int i=0; i<strlen(str); i++)
    {
        if(str[i]==' ')
        {
            index[ptr] = i-1;
            ptr++;

            index[ptr] = i+1;
            ptr++;

        
        }
    }

    index[ptr] = strlen(str)-1;

    for(int i=0; i<=ptr; i++)
    {
        printf("%d ", index[i]);
    }




    //////////////
    char * d_str;
    char * d_word;
    int count = 0;
    int *d_count;
    char word[10] = "hii";


    hipMalloc((void**)&d_str, strlen(str)*sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_str, str,strlen(str)*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count,sizeof(int),hipMemcpyHostToDevice);

    int num_words =  (ptr+1)/2;

   
    word_Counts<<<1, num_words>>>(d_str, index, )


}
