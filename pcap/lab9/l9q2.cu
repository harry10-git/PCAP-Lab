
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define matrix dimensions and kernel dimensions
#define WIDTH 4
#define HEIGHT 4
#define KERNEL_SIZE 3

// CUDA kernel for 2D convolution
__global__ void convolution2D(int *input, int *mask, int *output, int width, int height, int kernelSize)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width)
    {
        int sum = 0;
        int offset = kernelSize / 2;

        for (int i = 0; i < kernelSize; i++)
        {
            for (int j = 0; j < kernelSize; j++)
            {
                int r = row + i - offset;
                int c = col + j - offset;

                if (r >= 0 && r < height && c >= 0 && c < width)
                {
                    sum += input[r * width + c] * mask[i * kernelSize + j];
                }
            }
        }

        output[row * width + col] = sum;
    }
}

int main()
{
    int input[HEIGHT][WIDTH];                                               // Input matrix
    int mask[KERNEL_SIZE][KERNEL_SIZE] = {{1, 1, 1}, {1, 1, 1}, {1, 1, 1}}; // Kernel
    int output[HEIGHT][WIDTH];                                              // Output matrix

    int *d_input, *d_mask, *d_output;

    // Initialize input matrix (for simplicity)
    for (int i = 0; i < HEIGHT; i++)
    {
        for (int j = 0; j < WIDTH; j++)
        {
            input[i][j] = 1;
        }
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_input, WIDTH * HEIGHT * sizeof(int));
    hipMalloc((void **)&d_mask, KERNEL_SIZE * KERNEL_SIZE * sizeof(int));
    hipMalloc((void **)&d_output, WIDTH * HEIGHT * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_input, input, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, KERNEL_SIZE * KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid((WIDTH + 15) / 16, (HEIGHT + 15) / 16);
    dim3 dimBlock(16, 16);

    // Launch the CUDA kernel
    convolution2D<<<dimGrid, dimBlock>>>(d_input, d_mask, d_output, WIDTH, HEIGHT, KERNEL_SIZE);

    // Copy the result back to the host
    hipMemcpy(output, d_output, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);

    // Print the output matrix
    for (int i = 0; i < HEIGHT; i++)
    {
        for (int j = 0; j < WIDTH; j++)
        {
            printf("%d\t", output[i][j]);
        }
        printf("\n");
    }

    return 0;
}