/*2. Write a CUDA program that reads a string S and produces the string RS as follows:
Input string S: PCAP
Output string RS: PCAPPCAPCP
Note: Each work item copies required number of characters from S in RS.
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>



__global__ void stringCopy(char *d_s, char *d_rs, int n)
{
    int i = threadIdx.x;
    int j = 0;
    while (j < n)
    {
        d_rs[i*n + j] = d_s[j];
        j++;
    }
}

int main()
{
    char s[100], rs[100];
    char *d_s, *d_rs;
    int n;
    printf("Enter the string: ");
    scanf("%s", s);
    n = strlen(s);
    hipMalloc((void **)&d_s, n * sizeof(char));
    hipMalloc((void **)&d_rs, n * n * sizeof(char));
    hipMemcpy(d_s, s, n * sizeof(char), hipMemcpyHostToDevice);
    stringCopy<<<1, n>>>(d_s, d_rs, n);
    hipMemcpy(rs, d_rs, n * n * sizeof(char), hipMemcpyDeviceToHost);
    printf("Output string: %s\n", rs);
    hipFree(d_s);
    hipFree(d_rs);
    return 0;
}