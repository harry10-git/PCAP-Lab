#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 1024

__global__ void cudaCount(char * str, unsigned int *d_count)
{
    int tid = threadIdx.x;
    if(str[tid] == 'a'){
        atomicAdd(d_count,1);
    }
}

int main()
{
    char str[N];
    char *d_str;

    unsigned int *count, *d_count, *result;
    count = (unsigned int *)malloc(sizeof(unsigned int));
    result = (unsigned int *)malloc(sizeof(unsigned int));
    printf("enter the string: ");
    scanf("%s", str);

    hipMalloc((void **)&d_str, strlen(str)*sizeof(char));
    hipMalloc((void **)&d_count, sizeof(unsigned int));


    hipMemcpy(d_str, str, strlen(str)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, count, sizeof(unsigned int), hipMemcpyHostToDevice);

    cudaCount<<<1, strlen(str)>>>(d_str, d_count);
    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Total occurences of a=%u\n", *result);
    hipFree(d_str);
    hipFree(d_count);
    printf("\n");

    return 0;
}
