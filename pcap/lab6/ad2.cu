/*2) Write a CUDA program that takes a string Sin as input and one integer value N and produces
an output string , Sout, in parallel by concatenating input string Sin, N times as shown below.
Input:
Sin = “Hello” N = 3
Ouput:
Sout = “HelloHelloHello”
Note: Every thread copies the same character from the Input string S, N times to the re-
quired position.
*/

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void copystr(char *str, char * r_str, int len)
{
   int tid = threadIdx.x; int i; int pos = tid;


   for(i=0; i<len; i++)
   {
      r_str[pos] = str[tid];
      pos = pos + len;
   } 
}

int main(void)
{
   char str[100];
   printf("enter a string: ");
   scanf("%s", str);
   int len = sizeof(str)/sizeof(char);
   char *d_str, *d_rstr;
   hipMalloc((void **)&d_str, len*sizeof(char));
   hipMalloc((void **)&d_rstr, len*sizeof(char));
   hipMemcpy(d_str, str, len*sizeof(char), hipMemcpyHostToDevice);
   copystr<<<1, len>>>(d_str, d_rstr, len);
   char rstr[len];
   hipMemcpy(rstr, d_rstr, len*sizeof(char), hipMemcpyDeviceToHost);
   printf("output string: %s\n", rstr);
   hipFree(d_str);
   hipFree(d_rstr);
   return 0;
}