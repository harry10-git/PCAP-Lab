#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

// CUDA kernel to replace matrix elements
__global__ void replace(int* mat, int* out, int m, int n) {
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within the valid matrix range
    if (rid < m && cid < n) {
        int val = mat[rid * n + cid];
        int binary = 0, bitcount = 0, rev = 0;

        // Check if the element is on the border of the matrix
        if (rid == 0 || rid == m - 1 || cid == 0 || cid == n - 1) {
            rev = val; // If it is, set the reversed value to the original value
        } else {
            // Convert the value to binary and reverse it
            while (val > 0) {
                binary *= 10;
                if (val % 2 == 0)
                    binary += 1;
                val >>= 1;
                bitcount++;
            }

            while (bitcount--) {
                rev = rev * 10 + binary % 10;
                binary /= 10;
            }
        }

        // Store the reversed value in the output matrix
        out[rid * n + cid] = rev;
    }
}

int main() {
    int m, n;
    printf("Enter the dimensions of the matrix: ");
    scanf("%d %d", &m, &n);

    int sizemat = m * n * sizeof(int);
    int *mat = (int*)malloc(sizemat);
    int *out = (int*)malloc(sizemat);

    printf("\nEnter the matrix elements:\n");
    for (int i = 0; i < m * n; i++)
        scanf("%d", mat + i);

    int *d_mat, *d_out;
    hipMalloc((void**)&d_mat, sizemat);
    hipMalloc((void**)&d_out, sizemat);

    // Copy input matrix from host to device
    hipMemcpy(d_mat, mat, sizemat, hipMemcpyHostToDevice);

    // Define grid and block dimensions for CUDA kernel
    dim3 blockDim(32, 32); // 32x32 threads per block
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);

    // Launch the CUDA kernel
    replace<<<gridDim, blockDim>>>(d_mat, d_out, m, n);

    // Copy the result matrix from device to host
    hipMemcpy(out, d_out, sizemat, hipMemcpyDeviceToHost);

    // Print the resultant matrix
    printf("\nResultant Matrix:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++)
            printf("\t%d", out[i * n + j]);
        printf("\n");
    }

    // Free allocated memory
    hipFree(d_mat);
    hipFree(d_out);
    free(mat);
    free(out);

    return 0;
}
