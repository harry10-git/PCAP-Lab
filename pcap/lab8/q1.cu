/* Write a program in CUDA to perform parallel Sparse Matrix - Vector multiplication using com-
pressed sparse row (CSR) storage format. Represent the input sparse matrix in CSR format in the
host code. */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



// function to multiply sparse matrix and vector from below code
__global__ void sparse_matrix_vector_multiplication(int *row_ptr, int *col_index, int *vec, int *result, int n, int m)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k, l;
    if (i < n && j < m)
    {
        for (k = row_ptr[i]; k < row_ptr[i + 1]; k++)
        {
            l = col_index[k];
            result[i] += vec[l] * col_index[k];
        }
    }
}



int main(void)
{
    // take matrix input and convert it to csr
    int n, m, i, j, k, l, count = 0;
    printf("Enter the number of rows and columns of the matrix: ");
    scanf("%d %d", &n, &m);
    int *mat = (int *)malloc(n * m * sizeof(int));
    printf("Enter the matrix: ");
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < m; j++){
            scanf("%d", &mat[i * m + j]);
        }
    }

    // print the matrix
    printf("The matrix is: \n");
    for (i = 0; i < n; i++){
        for (j = 0; j < m; j++){
            printf("%d ", mat[i * m + j]);
        }
        printf("\n");
    }

    // convert to csr, calcluate col_index and row_ptr
    int *col_index = (int *)malloc(n * m * sizeof(int));
    int *row_ptr = (int *)malloc((n + 1) * sizeof(int));
    row_ptr[0] = 0;
    for (i = 0; i < n; i++){
        for (j = 0; j < m; j++){
            if (mat[i * m + j] != 0){
                col_index[count] = j;
                count++;
            }
        }
        row_ptr[i + 1] = count;
    }

    // display col_index and row_ptr
    printf("The col_index is: \n");
    for (i = 0; i < count; i++){
        printf("%d ", col_index[i]);
    }
    printf("\n");
    printf("The row_ptr is: \n");
    for (i = 0; i < n + 1; i++){
        printf("%d ", row_ptr[i]);
    }
    printf("\n");

    // take vector input
    int *vec = (int *)malloc(m * sizeof(int));
    printf("Enter the vector: ");
    for (i = 0; i < m; i++){
        scanf("%d", &vec[i]);
    }


    // launch kernel to multiply sparse matrix and vector
    int *result = (int *)malloc(n * sizeof(int));
    int *d_row_ptr, *d_col_index, *d_vec, *d_result;
    hipMalloc((void **)&d_row_ptr, (n + 1) * sizeof(int));
    hipMalloc((void **)&d_col_index, count * sizeof(int));
    hipMalloc((void **)&d_vec, m * sizeof(int));
    hipMalloc((void **)&d_result, n * sizeof(int));
    hipMemcpy(d_row_ptr, row_ptr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, col_index, count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec, m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, n * sizeof(int), hipMemcpyHostToDevice);
    dim3 block(32, 32);
    dim3 grid((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);
    sparse_matrix_vector_multiplication<<<grid, block>>>(d_row_ptr, d_col_index, d_vec, d_result, n, m);
    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    // print the result
    printf("The result is: \n");
    for (i = 0; i < n; i++){
        printf("%d ", result[i]);
    }
    printf("\n");
    
            


}