#include "hip/hip_runtime.h"
/*
Write a program in CUDA to read MXN matrix A and replace 1 st row of this matrix by same
elements, 2 nd row elements by square of each element and 3 rd row elements by cube of each element
and so on.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

__global__ void processMatrix(int *A, int *B, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols)
    {
        int index = row * numCols + col;
        int element = A[index];
        int power = row + 1; // Power for the current row

        if(power == 1)
        {
             B[index] = element;
        }

        // Replace elements based on the row number
        else if (power == 2)
        {
            B[index] = element * element; // Square for even rows
        }
        else if (power == 3)
        {
            B[index] = element * element * element; // Cube for odd rows
        }
    }
}

int main()
{
    int numRows = 3; // Number of rows
    int numCols = 4; // Number of columns

    // Input matrix A
    int A[numRows][numCols] = {
        {1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12}
    };

    // Allocate device memory for matrix A and B
    int *d_A, *d_B;
    hipMalloc((void **)&d_A, sizeof(int) * numRows * numCols);
    hipMalloc((void **)&d_B, sizeof(int) * numRows * numCols);

    // Copy matrix A from host to device
    hipMemcpy(d_A, A, sizeof(int) * numRows * numCols, hipMemcpyHostToDevice);

    // Define kernel launch configuration
    dim3 dimGrid((numCols + BLOCK_SIZE - 1) / BLOCK_SIZE, (numRows + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Launch kernel
    processMatrix<<<dimGrid, dimBlock>>>(d_A, d_B, numRows, numCols);

    // Copy the result matrix B from device to host
    int B[numRows][numCols];
    hipMemcpy(B, d_B, sizeof(int) * numRows * numCols, hipMemcpyDeviceToHost);

    // Print the result matrix B
    printf("Result Matrix B:\n");
    for (int i = 0; i < numRows; i++)
    {
        for (int j = 0; j < numCols; j++)
        {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}



