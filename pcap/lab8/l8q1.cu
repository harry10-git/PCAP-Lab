// Vector multiplication using com-pressed sparse row (CSR) storage format.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>       


int main(void)
{
    int row =3, col =3;
    int mat[3][3]= {{0,0,9}, {0,5,0}, {2,0,1}};


    // convert to csr
    int i,j;
    int col_index[row*col];
    int col_ptr =0;

    for(i=0; i<row; i++)
    {
        for(j=0; j<col; j++)
        {
            if(mat[i][j] != 0)
            {
                col_index[col_ptr++] = j; 
            }
        }
    }
    for(i=0; i<col_ptr; i++)
    {
        printf("%d ", col_index[i]);
    }
    printf("\n");

    // finding row_ptr
    int count =-1, flag = 0;
    int row_ptr[row];
    int row_index = 0;
    
    for(i=0; i<row; i++)
    {
        for(j=0; j<col ;j++)
        {
            if(mat[i][j] != 0)
            {
                count +=1;
            }

            if(flag == 0 && mat[i][j] !=0)
            {
                row_ptr[row_index++] = count;
                flag = 1;
            }
            
        }
        flag = 0;
    }

    for(i=0; i<row_index; i++)
    {
        printf("%d ", row_ptr[i]);
    }
    printf("\n");



}