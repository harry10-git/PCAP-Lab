#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

// Function to convert 2D matrix to CSR format
void convertToCSR(int **matrix, int numRows, int numCols, int **row_ptr, int **col_idx, int **values)
{
    int nnz = 0;
    for (int i = 0; i < numRows; i++)
    {
        for (int j = 0; j < numCols; j++)
        {
            if (matrix[i][j] != 0)
            {
                nnz++;
            }
        }
    }

    *row_ptr = (int *)malloc(sizeof(int) * (numRows + 1));
    *col_idx = (int *)malloc(sizeof(int) * nnz);
    *values = (int *)malloc(sizeof(int) * nnz);

    int index = 0;
    (*row_ptr)[0] = 0;
    for (int i = 0; i < numRows; i++)
    {
        for (int j = 0; j < numCols; j++)
        {
            if (matrix[i][j] != 0)
            {
                (*col_idx)[index] = j;
                (*values)[index] = matrix[i][j];
                index++;
            }
        }
        (*row_ptr)[i + 1] = index;
    }
}

__global__ void csrKernel(int *row_ptr, int *col_idx, int *values, int *x, int *y, int numRows)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numRows)
    {
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        int sum = 0;
        for (int i = row_start; i < row_end; i++)
        {
            int col = col_idx[i];
            sum += values[i] * x[col];
        }
        y[row] = sum;
    }
}

int main()
{
    int numRows = 3;
    int numCols = 3;

    // Input 2D matrix
    int **matrix = (int **)malloc(numRows * sizeof(int *));
    for (int i = 0; i < numRows; i++)
    {
        matrix[i] = (int *)malloc(numCols * sizeof(int));
    }
    // Example input matrix
    matrix[0][0] = 1;
    matrix[0][1] = 0;
    matrix[0][2] = 2;
    matrix[1][0] = 1;
    matrix[1][1] = 3;
    matrix[1][2] = 5;
    matrix[2][0] = 0;
    matrix[2][1] = 0;
    matrix[2][2] = 9;

    // Convert 2D matrix to CSR format
    int *row_ptr, *col_idx, *values;
    convertToCSR(matrix, numRows, numCols, &row_ptr, &col_idx, &values);

    // Display CSR format on host
    printf("CSR Format:\n");
    for (int i = 0; i <= numRows; i++)
    {
        printf("%d ", row_ptr[i]);
    }
    printf("\n");
    for (int i = 0; i < row_ptr[numRows]; i++)
    {
        printf("%d ", col_idx[i]);
    }
    printf("\n");
    for (int i = 0; i < row_ptr[numRows]; i++)
    {
        printf("%d ", values[i]);
    }
    printf("\n");

    // Input vector
    int x[] = {1, 2, 3};

    // Allocate device memory for CSR arrays and vector x
    int *d_row_ptr, *d_col_idx, *d_values, *d_x, *d_y;
    hipMalloc((void **)&d_row_ptr, sizeof(int) * (numRows + 1));
    hipMalloc((void **)&d_col_idx, sizeof(int) * row_ptr[numRows]);
    hipMalloc((void **)&d_values, sizeof(int) * row_ptr[numRows]);
    hipMalloc((void **)&d_x, sizeof(int) * numCols);
    hipMalloc((void **)&d_y, sizeof(int) * numRows);

    // Copy data from host to device
    hipMemcpy(d_row_ptr, row_ptr, sizeof(int) * (numRows + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx, sizeof(int) * row_ptr[numRows], hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, sizeof(int) * row_ptr[numRows], hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, sizeof(int) * numCols, hipMemcpyHostToDevice);

    // Define kernel launch configuration
    dim3 dimGrid((numRows + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    // Launch kernel
    csrKernel<<<dimGrid, dimBlock>>>(d_row_ptr, d_col_idx, d_values, d_x, d_y, numRows);

    // Copy the result back to host
    int y[numRows];
    hipMemcpy(y, d_y, sizeof(int) * numRows, hipMemcpyDeviceToHost);

    // Print the result
    printf("Result of Sparse Matrix-Vector Multiplication (CSR format):\n");
    for (int i = 0; i < numRows; i++)
    {
        printf("%d ", y[i]);
    }
    printf("\n");

    // Free host and device memory
    for (int i = 0; i < numRows; i++)
    {
        free(matrix[i]);
    }
    free(matrix);
    free(row_ptr);
    free(col_idx);
    free(values);
    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
