// Write a program in CUDA to process a 1D array containing angles in radians to generate
// sine of the angles in the output array. Use appropriate function.


#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void sine(float *a, float *b, int n)
{
    int id = threadIdx.x;
    if(id<n)
    {
        b[id] = sin(a[id]);
    }
}

int main(void)
{
    int n;
    float *a, *b;
    float *d_a, *d_b;
    printf("Enter the size of the array: ");
    scanf("%d", &n);
    a = (float *)malloc(n*sizeof(float));
    b = (float *)malloc(n*sizeof(float));

    //cuda malloc
    hipMalloc((void **)&d_a, n*sizeof(float));
    hipMalloc((void **)&d_b, n*sizeof(float));
   

    printf("Enter the elements of the array: ");
    for(int i=0; i<n; i++)
    {
       scanf("%f", &a[i]);
    }

    hipMemcpy(d_a, a, n*sizeof(float), hipMemcpyHostToDevice);
    // call the kernel
    sine<<<n, 1>>>(d_a, d_b, n);

    hipMemcpy(b, d_b, n*sizeof(float), hipMemcpyDeviceToHost);
    
    for(int i=0; i<n; i++)
    {
        printf("%f\n", b[i]);
    }
    return 0;


    return 0;
}