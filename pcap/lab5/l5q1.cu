// Write a program in CUDA to add two vectors of length N using
// a) block size as N  b) N threads

#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(void)
{
    int n;
    printf("enter value of n: ");
    scanf("%d",&n);

    int size = n * sizeof(int);
    int a[n], b[n], c[n];
    int *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

    printf("vector 1 : ");
    for(int i=0; i<n; i++)
    {
        scanf("%d",&a[i]);
    }
    printf("vector 2 : ");
    for(int i=0; i<n; i++)
    {
        scanf("%d",&b[i]);
    }
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    dim3 dimGrid(1,1,1);
    dim3 dimBlock(n,1,1);

    add<<<dimGrid,dimBlock>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    printf("sum : ");
    for(int i=0; i<n; i++)
    {  
        printf("%d ",c[i]);
    }
    hipFree(d_a);
    hipFree(d_b);  
    hipFree(d_c);
    return 0;
}
